#include "hip/hip_runtime.h"
/*
Author: Zhaodong Yang
Class: ECE6122
Last Date Modified: 2021/11/22
Description:
This file is to use cuda to calculate the interior temperature of a thin plate using Jocabi update.
reference: https://stackoverflow.com/questions/11994679/solving-2d-diffusion-heat-equation-with-cuda
https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial02/
https://stackoverflow.com/questions/7876624/timing-cuda-operations
*/


#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iomanip>
#include <unistd.h>

#define BLOCK_XDIM 32
#define BLOCK_YDIM 32

//Jacobi update using cuda
__global__ void Jacobi_Iterator(const double * __restrict__ T_old, double * __restrict__ T_new, const int N)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	int P = i + j * N;           // node (i,j)
	int up = i + (j + 1)*N;       // node (i,j+1)
	int down = i + (j - 1)*N;       // node (i,j-1)
	int right = (i + 1) + j * N;       // node (i+1,j) 
	int left = (i - 1) + j * N;       // node (i-1,j) 

	if (i > 0 && i < N - 1 && j>0 && j < N - 1) {
		T_new[P] = 0.25 * (T_old[right] + T_old[left] + T_old[up] + T_old[down]);
	}
}

//initialize of plate temperature
void Initialize(double * __restrict host_temperature, const int N)
{
	for (int j = 0; j < N; j++) {
		for (int i = 0;i < N; i ++){
			host_temperature[j * N + i] = 20.0;
		}
	}
	for ( int i = 0; i < N; i ++) {
		if (i > 0.3*(N-1) && i < 0.7*(N-1)) {
			host_temperature[i] = 100.0;
		}
	}
}

int main(int argc, char* argv[])
{
    int N;  
	int total_iteration;
	int c; 
	const char* optstring = "N:I:";
    while ((c = getopt(argc, argv, optstring)) != -1)
    {
        switch (c) {
        case 'N':
            //printf("opt is N, N is: %s\n", optarg);
            N = atoi(optarg) + 2;
			if (N <= 0 || !N){
				printf("Invalid parameters, please check your values.");
			}
            break;
        case 'I':
            //printf("opt is I, I is: %s\n", optarg);
            total_iteration = atoi(optarg);
			if (total_iteration <= 0 || !total_iteration){
				printf("Invalid parameters, please check your values.");
			}
            break;
        case '?':
            printf("error optopt: %c\n", optopt);
            printf("error opterr: %d\n", opterr);
            break;
        }
    }

	double *host_temperature = (double *)calloc(N * N, sizeof(double));
	double *host_temperature_old = (double *)calloc(N * N, sizeof(double));
	Initialize(host_temperature, N);
	Initialize(host_temperature_old, N);
	double *host_temperature_GPU_result = (double *)malloc(N * N * sizeof(double));

	double *gpu_temperature;     
	hipMalloc((void**)&gpu_temperature, N * N * sizeof(double));
	double *gpu_temperature_old; 
	hipMalloc((void**)&gpu_temperature_old, N * N * sizeof(double));

	hipMemcpy(gpu_temperature, host_temperature, N * N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_temperature_old, gpu_temperature, N * N * sizeof(double), hipMemcpyDeviceToDevice);

	dim3 dimBlock(BLOCK_XDIM, BLOCK_YDIM);
	dim3 dimGrid((N + BLOCK_XDIM) / BLOCK_XDIM, (N + BLOCK_YDIM) / BLOCK_YDIM);

	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < total_iteration; i = i + 2) {
		Jacobi_Iterator << <dimGrid, dimBlock, 0 >> > (gpu_temperature, gpu_temperature_old, N);   // --- Update gpu_temperature_old     starting from data stored in gpu_temperature
		Jacobi_Iterator << <dimGrid, dimBlock, 0 >> > (gpu_temperature_old, gpu_temperature, N);   // --- Update gpu_temperature         starting from data stored in gpu_temperature_old
	}

	hipMemcpy(host_temperature_GPU_result, gpu_temperature, N * N * sizeof(double), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time to generate:  %.2lf ms \n", time);
	
	std::ofstream outFile;
	outFile.open("finalTemperatures.csv");

	outFile.setf(std::ios::fixed);

	for (int j = 0; j < N; j ++){
		for (int i = 0; i < N; i ++){
			outFile <<std::setprecision(8) << host_temperature_GPU_result[j * N + i] << ',';
		}
		outFile << '\n';
	}
	
	free(host_temperature);
	free(host_temperature_GPU_result);

	hipFree(gpu_temperature);
	hipFree(gpu_temperature_old);

	return 0;
}